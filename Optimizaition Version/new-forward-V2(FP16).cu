#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include "./gpu-new-forward.h"

#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include ""
#include "hip/hip_fp16.h"

#define BLOCK_SIZE 16
#define KernelSize 7
#define InputfeatureMax 4
#define OutputfeatureMax 16
// __constant__ half Half_ConstK[OutputfeatureMax * InputfeatureMax * KernelSize * KernelSize];


__global__ void conv_forward_kernel(half* y, const half* x, const half* k, const int B, const int M, const int C, const int H, const int W, const int K)
{
	/*
	Modify this function to implement the forward pass described in Chapter 16.
	We have added an additional dimension to the tensors to support an entire mini-batch
	The goal here is to be correct AND fast.

	Function paramter definitions:
	y - output
	x - input
	k - kernel
	B - batch_size (number of images in x)
	M - number of output feature maps
	C - number of input feature maps
	H - input height dimension
	W - input width dimension
	K - kernel height and width (K x K) = 7
	*/
	const int H_out = H - K + 1;
	const int W_out = W - K + 1;

	// We have some nice #defs for you below to simplify indexing. Feel free to use them, or create your own.
	// An example use of these macros:
	// float a = y4d(0,0,0,0)
	// y4d(0,0,0,0) = a

#define x4d(i3, i2, i1, i0) x[(i3) * (C * H * W) + (i2) * (H * W) + (i1) * (W) + i0]
#define y4d(i3, i2, i1, i0) y[(i3) * (M * H_out * W_out) + (i2) * (H_out * W_out) + (i1) * (W_out) + i0]
#define k4d(i3, i2, i1, i0) k[(i3) * (C * K * K) + (i2) * (K * K) + (i1) * (K) + i0]

	int W_grid = ceil(W_out / (1.0 * blockDim.x));
	//int H_grid = ceil(1.0 * W_out / blockDim.y);
	// Insert your GPU convolution kernel code here
	// the n simple
	int n = blockIdx.x;
	// the m_th output feature
	int m = blockIdx.y;
	// the absolute index
	int h = (blockIdx.z / W_grid) * BLOCK_SIZE + threadIdx.y;
	int w = (blockIdx.z % W_grid) * BLOCK_SIZE + threadIdx.x;
	if (w < W_out && h < H_out)
	{
		half acc = 0.0;
		for (int c = 0; c < C; c++)
		{
			for (int p = 0; p < K; p++)
				for (int q = 0; q < K; q++)
					acc = __hadd(__hmul(x4d(n, c, h + p, w + q), k4d(m, c, p, q)), acc);
		}
		y4d(n, m, h, w) = acc;
	}
#undef x4d
#undef y4d
#undef k4d
}

__global__ void float2half_mat(const float* float_array, half* half_array, int length)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	for (int idx = index; idx < length; idx += gridDim.x * blockDim.x)
	{
		half_array[idx] = __float2half(float_array[idx]);
	}
	return;
}

__global__ void half2float_mat(half* half_array, float* float_array, int length)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	for (int idx = index; idx < length; idx += gridDim.x * blockDim.x)
	{
		float_array[idx] = __half2float(half_array[idx]);
	}
	return;
}



__host__ void GPUInterface::conv_forward_gpu_prolog(float* host_y, float* host_x, float* host_k, float** device_y_ptr, float** device_x_ptr, float** device_k_ptr, const int B, const int M, const int C, const int H, const int W, const int K)
{
	// Allocate memory and copy over the relevant data structures to the GPU
	int H_out = H - K + 1;
	int W_out = W - K + 1;
	hipMalloc(device_x_ptr, sizeof(float) * B * C * H * W);
	hipMalloc(device_y_ptr, sizeof(float) * B * M * W_out * H_out);
	hipMalloc(device_k_ptr, sizeof(float) * M * C * K * K);
	hipMemcpy(*device_x_ptr, host_x, sizeof(float) * B * C * H * W, hipMemcpyHostToDevice);
	hipMemcpy(*device_k_ptr, host_k, sizeof(float) * M * C * K * K, hipMemcpyHostToDevice);
	//hipMemcpyToSymbol(HIP_SYMBOL(ConstK), host_k, sizeof(float) * M * C * K * K);
	// We pass double pointers for you to initialize the relevant device pointers,
	//  which are passed to the other two functions.

	// Useful snippet for error checking
	// hipError_t error = hipGetLastError();
	// if(error != hipSuccess)
	// {
	//     std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
	//     exit(-1);
	// }

	// input unroll
}



// to see acceleration, using nsys statistic in ms form of kernel call
__host__ void GPUInterface::conv_forward_gpu(float* device_y, const float* device_x, const float* device_k, const int B, const int M, const int C, const int H, const int W, const int K)
{
	// Set the kernel dimensions and call the kernel
	int H_out = H - K + 1;
	int W_out = W - K + 1;
	int W_grid = ceil(W_out / (1.0 * BLOCK_SIZE));
	int H_grid = ceil(H_out / (1.0 * BLOCK_SIZE));
	int Z = W_grid * H_grid;

	half* half_device_k;
	half* half_device_x;
	half* half_device_y;

	hipMalloc(&half_device_x, sizeof(float) / 2 * B * C * H * W);
	hipMalloc(&half_device_y, sizeof(float) / 2 * B * M * W_out * H_out);
	hipMalloc(&half_device_k, sizeof(float) / 2 * M * C * K * K);

	int Xlength = B * C * H * W;
	int Ylength = B * M * W_out * H_out;
	int Klength = M * C * K * K;


	dim3 Half_gridDim(16, 1, 1);
	dim3 Half_blockDim(1024, 1, 1);
	float2half_mat << < Half_gridDim, Half_blockDim >> > (device_x, half_device_x, Xlength);
	hipDeviceSynchronize();
	float2half_mat << < Half_gridDim, Half_blockDim >> > (device_k, half_device_k, Klength);
	hipDeviceSynchronize();

	//hipMemcpyToSymbol(HIP_SYMBOL(Half_ConstK), half_device_k, sizeof(half) * M * C * K * K);
	dim3 grid_Dim(B, M, Z);
	dim3 block_Dim(BLOCK_SIZE, BLOCK_SIZE, 1);
	conv_forward_kernel << < grid_Dim, block_Dim >> > (half_device_y, half_device_x, half_device_k, B, M, C, H, W, K);
	hipDeviceSynchronize();

	half2float_mat << < Half_gridDim, Half_blockDim >> > (half_device_y, device_y, Ylength);
	hipDeviceSynchronize();

	hipFree(half_device_x);
	hipFree(half_device_y);
	hipFree(half_device_k);


}


__host__ void GPUInterface::conv_forward_gpu_epilog(float* host_y, float* device_y, float* device_x, float* device_k, const int B, const int M, const int C, const int H, const int W, const int K)
{
	// Copy the output back to host
	int H_out = H - K + 1;
	int W_out = W - K + 1;
	hipMemcpy(host_y, device_y, sizeof(float) * B * M * W_out * H_out, hipMemcpyDeviceToHost);
	// Free device memory
	hipFree(device_k);
	hipFree(device_x);
	hipFree(device_y);
}


__host__ void GPUInterface::get_device_properties()
{
	int deviceCount;
	hipGetDeviceCount(&deviceCount);

	for (int dev = 0; dev < deviceCount; dev++)
	{
		hipDeviceProp_t deviceProp;
		hipGetDeviceProperties(&deviceProp, dev);

		std::cout << "Device " << dev << " name: " << deviceProp.name << std::endl;
		std::cout << "Computational capabilities: " << deviceProp.major << "." << deviceProp.minor << std::endl;
		std::cout << "Max Global memory size: " << deviceProp.totalGlobalMem << std::endl;
		std::cout << "Max Constant memory size: " << deviceProp.totalConstMem << std::endl;
		std::cout << "Max Shared memory size per block: " << deviceProp.sharedMemPerBlock << std::endl;
		std::cout << "Max threads per block: " << deviceProp.maxThreadsPerBlock << std::endl;
		std::cout << "Max block dimensions: " << deviceProp.maxThreadsDim[0] << " x, " << deviceProp.maxThreadsDim[1] << " y, " << deviceProp.maxThreadsDim[2] << " z" << std::endl;
		std::cout << "Max grid dimensions: " << deviceProp.maxGridSize[0] << " x, " << deviceProp.maxGridSize[1] << " y, " << deviceProp.maxGridSize[2] << " z" << std::endl;
		std::cout << "Warp Size: " << deviceProp.warpSize << std::endl;
	}
}
