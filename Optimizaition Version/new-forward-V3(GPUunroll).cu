#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include "./gpu-new-forward.h"

#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include ""


#define BLOCK_SIZE 16
#define ImageNum 20


//__constant__ float ConstK[OutputfeatureMax * InputfeatureMax * KernelSize * KernelSize];
//__constant__ float Half_ConstK[OutputfeatureMax * InputfeatureMax * KernelSize * KernelSize];


__global__ void conv_forward_kernel(float* device_y, float* unroll_x, const float* device_k,
	const int B, const int M, const int C, const int H, const int W, const int K, const int sampleID)
{
	/*
	Modify this function to implement the forward pass described in Chapter 16.
	We have added an additional dimension to the tensors to support an entire mini-batch
	The goal here is to be correct AND fast.

	Function paramter definitions:
	y - output
	x - input
	k - kernel
	B - batch_size (number of images in x)
	M - number of output feature maps
	C - number of input feature maps
	H - input height dimension
	W - input width dimension
	K - kernel height and width (K x K) = 7
	*/
	const int H_out = H - K + 1;
	const int W_out = W - K + 1;

	// We have some nice #defs for you below to simplify indexing. Feel free to use them, or create your own.
	// An example use of these macros:
	// float a = y4d(0,0,0,0)
	// y4d(0,0,0,0) = a

	__shared__ float k_s[BLOCK_SIZE][BLOCK_SIZE];
	__shared__ float x_s[ImageNum][BLOCK_SIZE][BLOCK_SIZE];
	int output_x = blockIdx.x * blockDim.x + threadIdx.x;
	int output_y = blockIdx.y * blockDim.y + threadIdx.y;
	int tx, ty;
	int numArow = M;
	int numAcolumn = C * K * K;
	int numBrow = numAcolumn;
	int numBcolumn = W_out * H_out;
	int relative_pos = blockIdx.z;


	tx = threadIdx.x;
	ty = threadIdx.y;
	float c_values = 0;
	for (int i = 0; i < ((numAcolumn - 1) / BLOCK_SIZE + 1); i++)
	{
		if (i * BLOCK_SIZE + tx < numAcolumn && output_y < numArow)
		{
			k_s[ty][tx] = device_k[output_y * numAcolumn + i * BLOCK_SIZE + tx];
		}
		else
		{
			k_s[ty][tx] = 0;
		}
		if (i * BLOCK_SIZE + ty < numBrow && output_x < numBcolumn && (sampleID + relative_pos) < B)
		{
			x_s[relative_pos][ty][tx] = unroll_x[relative_pos * (numBcolumn * numBrow) + (i * BLOCK_SIZE + ty) * numBcolumn + output_x];
		}
		else
		{
			x_s[relative_pos][ty][tx] = 0;
		}
		__syncthreads();
		for (int i = 0; i < BLOCK_SIZE; i++)
		{
			c_values += k_s[ty][i] * x_s[relative_pos][i][tx];
		}
		__syncthreads();
	}
	if (output_y < numArow && output_x < numBcolumn && (sampleID + relative_pos) < B)
	{
		device_y[(sampleID + relative_pos) * (M * W_out * H_out) + output_y * (W_out * H_out) + output_x] = c_values;
	}

}

__global__ void Input_matrix_unroll(const float* device_x, float* unrolled_x,
	int C, int K, int W, int H, int sampleID)
{
	const int W_out = W - K + 1;
	const int H_out = H - K + 1;

#define x4d(i3, i2, i1, i0) device_x[(i3) * (C * H * W) + (i2) * (H * W) + (i1) * (W) + (i0)]
#define unrollx3d(i2,i1,i0) unrolled_x[(i2) * (C*K*K*W_out*H_out) + (i1) * ( W_out * H_out ) + (i0)]
	int w = blockIdx.x * blockDim.x + threadIdx.x;
	int h = blockIdx.y * blockDim.y + threadIdx.y;
	int n = blockIdx.z + sampleID;
	if (w < W_out & h < H_out)
	{
		for (int c = 0; c < C; c++)
		{
			for (int p = 0; p < K; p++)
			{
				for (int q = 0; q < K; q++)
				{
					unrollx3d(blockIdx.z, (c * K * K) + (p * K) + q, h * W_out + w) = x4d(n, c, h + p, w + q);
				}
			}
		}
	}
#undef x4d
#undef unrollx3d
}

// dim3 HMU_gridDim(ceil,ceil,M)
// dim3 HMU_blockDim(BLOCK_SIZE,BLOCK_SIZE,1)

//__global__ void Kernel_unroll(const float* device_k, float* unrolled_k,
//	int C, int K, int M)
//{
//#define k4d(i3, i2, i1, i0) device_k[(i3) * (C * K * K) + (i2) * (K * K) + (i1) * (K) + i0]
//#define unrollk2d(i1,i0) unrolled_k[ (i1) * (C*K*K) + (i0)]
//	int idx_x = blockIdx.x * blockDim.x + threadIdx.x;
//	int idx_y = blockIdx.y * blockDim.y + threadIdx.y;
//	int c = blockIdx.z;
//	if (idx_x < K && idx_y < K)
//	{
//		for (int m = 0; m < M; m++)
//		{
//			unrollk2d(m, c * (K * K) + idx_y * K + idx_x) = k4d(m, c, idx_y, idx_x);
//		}
//	}
//#undef k4d
//#undef unrollk2d
//}




//__global__ void FeatureMap_mat_roll(float* device_y, float* unrolled_y,
//	int M, int W_out, int H_out)
//{
//#define y4d(i3, i2, i1, i0) device_y[(i3) * (M * H_out * W_out) + (i2) * (H_out * W_out) + (i1) * (W_out) + i0]
//#define unrolly3d(i2,i1,i0)	unrolled_y[(i2)*(M*W_out*H_out) + (i1)*(W_out*H_out) + (i0)]
//	int w = blockIdx.x * blockDim.x + threadIdx.x;
//	int h = blockIdx.y * blockDim.y + threadIdx.y;
//	int n = blockIdx.z;
//	if (w < W_out && h < H_out)
//	{
//		for (int m = 0; m < M; m++)
//		{
//			y4d(n, m, h, w) = unrolly3d(n, m, h * W_out + w);
//		}
//	}
//#undef y4d
//#undef unrolly2d
//	return;
//}


__host__ void GPUInterface::conv_forward_gpu_prolog(float* host_y, float* host_x, float* host_k, float** device_y_ptr, float** device_x_ptr, float** device_k_ptr, const int B, const int M, const int C, const int H, const int W, const int K)
{
	// Allocate memory and copy over the relevant data structures to the GPU
	int H_out = H - K + 1;
	int W_out = W - K + 1;
	hipMalloc(device_x_ptr, sizeof(float) * B * C * H * W);
	hipMalloc(device_y_ptr, sizeof(float) * B * M * W_out * H_out);
	hipMalloc(device_k_ptr, sizeof(float) * M * C * K * K);
	hipMemcpy(*device_x_ptr, host_x, sizeof(float) * B * C * H * W, hipMemcpyHostToDevice);
	hipMemcpy(*device_k_ptr, host_k, sizeof(float) * M * C * K * K, hipMemcpyHostToDevice);
	// We pass double pointers for you to initialize the relevant device pointers,
	//  which are passed to the other two functions.

	// Useful snippet for error checking
	// hipError_t error = hipGetLastError();
	// if(error != hipSuccess)
	// {
	//     std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
	//     exit(-1);
	// }

	// input unroll
}



// to see acceleration, using nsys statistic in ms form of kernel call
__host__ void GPUInterface::conv_forward_gpu(float* device_y, const float* device_x, const float* device_k, const int B, const int M, const int C, const int H, const int W, const int K)
{
	// Set the kernel dimensions and call the kernel
	int H_out = H - K + 1;
	int W_out = W - K + 1;
	int Numphoto = 25;

	float* unroll_device_k;
	float* unroll_device_x;
	float* unroll_device_y;

	// hipMalloc(&unroll_device_k, sizeof(float) * C * K * K * M);
	hipMalloc(&unroll_device_x, sizeof(float) * ImageNum * C * K * K * W_out * H_out);
	// hipMalloc(&unroll_device_y, sizeof(float) * B * M * W_out * H_out);

	/*std::cout << "value of K " << K << std::endl;
	std::cout << "value of C " << C << std::endl;
	std::cout << "value of H " << H << std::endl;
	std::cout << "value of W " << W << std::endl;
	std::cout << "value of M " << M << std::endl;*/

	dim3 IMU_gridDim(ceil(W_out / (1.0 * BLOCK_SIZE)), ceil(H_out / (1.0 * BLOCK_SIZE)), ImageNum);
	dim3 IMU_blockDim(BLOCK_SIZE, BLOCK_SIZE, 1);


	// dim3 KU_gridDim(ceil(K / (1.0 * BLOCK_SIZE)), ceil(K / (1.0 * BLOCK_SIZE)), M);
	// dim3 KU_blockDim(BLOCK_SIZE, BLOCK_SIZE,1);
	// Kernel_unroll << < KU_gridDim, KU_blockDim >> > (device_k, unroll_device_k, C, K, M);
	// hipDeviceSynchronize();

	dim3 CFK_gridDim(ceil((W_out * H_out) / (1.0 * BLOCK_SIZE)), ceil(M / (1.0 * BLOCK_SIZE)), ImageNum);
	dim3 CFK_blockDim(BLOCK_SIZE, BLOCK_SIZE, 1);
	for (int sampleID = 0; sampleID < B; sampleID += ImageNum)
	{
		Input_matrix_unroll << < IMU_gridDim, IMU_blockDim >> > (device_x, unroll_device_x, C, K, W, H, sampleID);
		hipDeviceSynchronize();
		conv_forward_kernel << < CFK_gridDim, CFK_blockDim >> > (device_y, unroll_device_x, device_k, B, M, C, H, W, K, sampleID);
		hipDeviceSynchronize();
	}

	hipFree(unroll_device_x);
	// dim3 FMR_gridDim(ceil(W_out * H_out / (1.0 * BLOCK_SIZE)), ceil(M / (1.0 * BLOCK_SIZE)), B);
	// dim3 FMR_blockDim(BLOCK_SIZE, BLOCK_SIZE, 1);
	// FeatureMap_mat_roll << < FMR_gridDim, FMR_blockDim >> > (device_y, unroll_device_y, M, W_out, H_out);
	// hipDeviceSynchronize();


	//hipFree(unroll_device_y);
	//hipFree(unroll_device_k);


}


__host__ void GPUInterface::conv_forward_gpu_epilog(float* host_y, float* device_y, float* device_x, float* device_k, const int B, const int M, const int C, const int H, const int W, const int K)
{
	// Copy the output back to host
	int H_out = H - K + 1;
	int W_out = W - K + 1;
	hipMemcpy(host_y, device_y, sizeof(float) * B * M * W_out * H_out, hipMemcpyDeviceToHost);
	// Free device memory
	hipFree(device_k);
	hipFree(device_x);
	hipFree(device_y);
}


__host__ void GPUInterface::get_device_properties()
{
	int deviceCount;
	hipGetDeviceCount(&deviceCount);

	for (int dev = 0; dev < deviceCount; dev++)
	{
		hipDeviceProp_t deviceProp;
		hipGetDeviceProperties(&deviceProp, dev);

		std::cout << "Device " << dev << " name: " << deviceProp.name << std::endl;
		std::cout << "Computational capabilities: " << deviceProp.major << "." << deviceProp.minor << std::endl;
		std::cout << "Max Global memory size: " << deviceProp.totalGlobalMem << std::endl;
		std::cout << "Max Constant memory size: " << deviceProp.totalConstMem << std::endl;
		std::cout << "Max Shared memory size per block: " << deviceProp.sharedMemPerBlock << std::endl;
		std::cout << "Max threads per block: " << deviceProp.maxThreadsPerBlock << std::endl;
		std::cout << "Max block dimensions: " << deviceProp.maxThreadsDim[0] << " x, " << deviceProp.maxThreadsDim[1] << " y, " << deviceProp.maxThreadsDim[2] << " z" << std::endl;
		std::cout << "Max grid dimensions: " << deviceProp.maxGridSize[0] << " x, " << deviceProp.maxGridSize[1] << " y, " << deviceProp.maxGridSize[2] << " z" << std::endl;
		std::cout << "Warp Size: " << deviceProp.warpSize << std::endl;
	}
}
